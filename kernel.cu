#include "hip/hip_runtime.h"
﻿#define BUILD_CUDA_GRAYSCALE

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <string>

#include "GrayScaleCuda.h"
#include "Pixel.h"

__global__ void grayScaleKernel(Pixel_t* image) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    Pixel_t inpixel = *(image + i);
    int value = (0.299 * inpixel.R) + (0.587 * inpixel.G) + (0.114 * inpixel.B);
    inpixel.R = value;
    inpixel.G = value;
    inpixel.B = value;
    *(image + i) = inpixel;
}

extern "C" EXP_CUDA_GRAYSCALE bool grayScaleWithCuda(std::string file_src, std::string file_dst)
{
	Pixel_t* dev_image = 0;
	hipError_t cudaStatus;

	/*
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_image, pixels_to_process * sizeof(Pixel_t));
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_image, image, pixels_to_process * sizeof(Pixel_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	int blockSize, gridSize;
	blockSize = 256;
	gridSize = (int)ceil((float)pixels_to_process / blockSize);

	grayScaleKernel<<<gridSize,blockSize>>>(dev_image);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMemcpy(image, dev_image, pixels_to_process * sizeof(Pixel_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	return true;

Error:
	hipFree(dev_image); */
	return false;
}
