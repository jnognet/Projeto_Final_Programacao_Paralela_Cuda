#include "hip/hip_runtime.h"
﻿#define BUILD_CUDA_GRAYSCALE

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

#include "GrayScaleCuda.h"

__global__ void grayScaleKernel(Pixel_t* image) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    Pixel_t inpixel = *(image + i);
    int value = (0.299 * inpixel.R) + (0.587 * inpixel.G) + (0.114 * inpixel.B);
    inpixel.R = value;
    inpixel.G = value;
    inpixel.B = value;
    *(image + i) = inpixel;
}

extern "C" EXP_CUDA_GRAYSCALE bool grayScaleWithCuda(Pixel_t* image, int pixels_to_process)
{
	Pixel_t* dev_image = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_image, pixels_to_process * sizeof(Pixel_t));
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_image, image, pixels_to_process * sizeof(Pixel_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	int blockSize, gridSize;
	blockSize = 256;
	gridSize = (int)ceil((float)pixels_to_process / blockSize);

	grayScaleKernel<<<gridSize,blockSize>>>(dev_image);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMemcpy(image, dev_image, pixels_to_process * sizeof(Pixel_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	return true;

Error:
	hipFree(dev_image);
	return false;
}
