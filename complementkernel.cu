#include "hip/hip_runtime.h"
﻿#pragma once

#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#include "CudaKernel.h"
#include "Pixel.h"

__global__ void complementKernel(Pixel_t* image) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	Pixel_t inpixel = *(image + i);
	inpixel.R = 255 - inpixel.R;
	inpixel.G = 255 - inpixel.G;
	inpixel.B = 255 - inpixel.B;
	*(image + i) = inpixel;
}

bool complementCuda(Pixel_t* image, int pixels_to_process)
{	
	hipError_t cudaStatus;
	Pixel_t* dev_image;	
	
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_image, pixels_to_process * sizeof(Pixel_t));
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_image, image, pixels_to_process * sizeof(Pixel_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	int blockSize, gridSize;
	blockSize = 256;
	gridSize = (int)ceil((float)pixels_to_process / blockSize);

	complementKernel<<<gridSize,blockSize>>>(dev_image);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMemcpy(image, dev_image, pixels_to_process * sizeof(Pixel_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		goto Error;
	}
		
	return true;

Error:
	hipFree(dev_image);
	return false;
}
