#include "hip/hip_runtime.h"
﻿#pragma once

#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#include "CudaKernel.h"
#include "Pixel.h"

__global__ void grayScaleKernel(Pixel_t* image) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	Pixel_t inpixel = *(image + i);
	int value = (0.299 * inpixel.R) + (0.587 * inpixel.G) + (0.114 * inpixel.B);
	inpixel.R = value;
	inpixel.G = value;
	inpixel.B = value;
	*(image + i) = inpixel;
}

bool grayScaleCuda(Pixel_t* image, int pixels_to_process)
{	
	hipError_t cudaStatus;
	Pixel_t* dev_image;	
	
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_image, pixels_to_process * sizeof(Pixel_t));
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_image, image, pixels_to_process * sizeof(Pixel_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	int blockSize, gridSize;
	blockSize = 256;
	gridSize = (int)ceil((float)pixels_to_process / blockSize);

	grayScaleKernel<<<gridSize,blockSize>>>(dev_image);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipMemcpy(image, dev_image, pixels_to_process * sizeof(Pixel_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		goto Error;
	}
		
	return true;

Error:
	hipFree(dev_image);
	return false;
}
